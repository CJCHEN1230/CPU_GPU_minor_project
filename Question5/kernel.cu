
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include<iostream>
using namespace std;
int main()
{
	//r value
	double r01=  1;
	double r24 = 1;	
	double r57 = 1;
	double r16 = 5;
	double r25 = 1;
	double r78 = 2;
	double r12 = 1;
	double r56 = 3;
	double r89 = 3;
	double r03 = 5;
	double r3e = 1;
	double r69 = 3;
	double r34 = 3;
	double r47 = 1;
	double r8a = 1;
	double rab = 3;
	double rad = 2;
	double reb = 2;
	double rbc = 2;
	double rdc = 1;
	double rdg = 5;
	double r9g = 1;
	double rgh = 1;
	double ref = 5;
	double rcf = 2;
	double rfh = 2;
	double rS0 = 1;
	double rEA = 1;
	double rEB = 1;


	//Q flow rate

	double Qin = 2.09457;

	double Q01 = Qin / 2.0;
	double Q24 = Qin / 8.0;
	double Q57 = Qin / 8.0;
	double Q16 = Qin / 4.0;
	double Q25 = Qin / 8.0;
	double Q78 = Qin / 4.0;
	double Q12 = Qin / 4.0;
	double Q56 = 0;
	double Q89 = 0;
	double Q03 = Qin / 2.0;
	double Q3e = Qin / 2.0;
	double Q69 = Qin / 4.0;
	double Q34 = 0;
	double Q47 = Qin / 8.0;
	double Q8a = Qin / 4.0;
	double Qab = Qin / 8.0;
	double Qad = Qin / 8.0;
	double Qeb = 0;
	double Qbc = Qin / 8.0;
	double Qdc = 0;
	double Qdg = Qin / 8.0;
	double Q9g = 0;
	double Qgh = Qin / 8.0;
	double Qef = Qin / 2.0;
	double Qcf = Qin / 8.0;
	double Qfh = Qin * 5.0 / 8.0;
	double QS0 = Qin;
	double QEA = Qin / 4.0;
	double QEB = Qin * 3.0 / 4.0;



	double dQ_1 = 0;
	double dQ_2 = 0;
	double dQ_3 = 0;
	double dQ_4 = 0;
	double dQ_5 = 0;
	double dQ_6 = 0;
	double dQ_7 = 0;
	double dQ_8 = 0;
	double dQ_9 = 0;
	double ds1 = 0;
	double ds2 = 0;

	int c01 = 1;
	int c24 = 1;
	int c57 = 1;
	int c16 = 1;
	int c25 = 1;
	int c78 = 1;
	int c12 = 1;
	int c56 = 1;
	int c89 = 1;
	int c03 = 1;
	int c3e = 1;
	int c69 = 1;
	int c34 = 1;
	int c47 = 1;

	int c8a = 1;
	int cab = 1;
	int cad = 1;
	int ceb = 1;
	int cbc = 1;
	int cdc = 1;
	int cdg = 1;
	int c9g = 1;
	int cgh = 1;
	int cef = 1;
	int ccf = 1;
	int cfh = 1;

	int cS0 = 1;
	int cEA = 1;
	int cEB = 1;


	int no_iteraion = 10000;
	


	for (int i = 0; i < no_iteraion; i++)
	{		
		c01 = Q01 >= 0 ? 1 : -1;
		c24 = Q24 >= 0 ? 1 : -1;
		c57 = Q57 >= 0 ? 1 : -1;
		c16 = Q16 >= 0 ? 1 : -1;
		c25 = Q25 >= 0 ? 1 : -1;
		c78 = Q78 >= 0 ? 1 : -1;
		c12 = Q12 >= 0 ? 1 : -1;
		c56 = Q56 >= 0 ? 1 : -1;
		c89 = Q89 >= 0 ? 1 : -1;
		c03 = Q03 >= 0 ? 1 : -1;
		c3e = Q3e >= 0 ? 1 : -1;
		c69 = Q69 >= 0 ? 1 : -1;
		c34 = Q34 >= 0 ? 1 : -1;
		c47 = Q47 >= 0 ? 1 : -1;

		c8a = Q8a >= 0 ? 1 : -1;
		cab = Qab >= 0 ? 1 : -1;
		cad = Qad >= 0 ? 1 : -1;
		ceb = Qeb >= 0 ? 1 : -1;
		cbc = Qbc >= 0 ? 1 : -1;
		cdc = Qdc >= 0 ? 1 : -1;
		cdg = Qdg >= 0 ? 1 : -1;
		c9g = Q9g >= 0 ? 1 : -1;
		cgh = Qgh >= 0 ? 1 : -1;
		cef = Qef >= 0 ? 1 : -1;
		ccf = Qcf >= 0 ? 1 : -1;
		cfh = Qfh >= 0 ? 1 : -1;

		cS0 = QS0 >= 0 ? 1 : -1;
		cEA = QEA >= 0 ? 1 : -1;
		cEB = QEB >= 0 ? 1 : -1;

		 dQ_1 = -(c01*r01 *Q01 *Q01 + c12*r12*Q12*Q12 + c24*r24*Q24*Q24 - c34*r34*Q34*Q34 - c03*r03*Q03*Q03)
			 / (2 * r01*fabs(Q01) + 2 * r12*fabs(Q12) + 2 * r24*fabs(Q24) + 2 * r34*fabs(Q34) + 2 * r03*fabs(Q03));

		 dQ_2 = -(c16*r16 *Q16 *Q16 - c56*r56*Q56*Q56 - c25*r25*Q25*Q25 - c12*r12*Q12*Q12)
			 / (2 * r16*fabs(Q16) + 2 * r56*fabs(Q56) + 2 * r25*fabs(Q25) + 2 * r12*fabs(Q12));

		 dQ_3 = -(c25*r25 *Q25 *Q25 + c57*r57*Q57*Q57 - c47*r47*Q47*Q47 - c24*r24*Q24*Q24)
			 / (2 * r25*fabs(Q25) + 2 * r57*fabs(Q57) + 2 * r47*fabs(Q47) + 2 * r24*fabs(Q24));

		 dQ_4 = -(c34*r34 *Q34 *Q34 + c47*r47*Q47*Q47 + c78*r78*Q78*Q78 + c8a*r8a*Q8a*Q8a + cab*rab*Qab*Qab - ceb*reb*Qeb*Qeb - c3e*r3e*Q3e*Q3e)
			 / (2 * r34*fabs(Q34) + 2 * r47*fabs(Q47) + 2 * r78*fabs(Q78) + 2 * r8a*fabs(Q8a) + 2 * rab*fabs(Qab) + 2 * reb*fabs(Qeb) + 2 * r3e*fabs(Q3e));

		 dQ_5 = -(c56*r56 *Q56 *Q56 + c69*r69*Q69*Q69 - c89*r89*Q89*Q89 - c78*r78*Q78*Q78 - c57*r57*Q57*Q57)
			 / (2 * r56*fabs(Q56) + 2 * r12*fabs(Q12) + 2 * r89*fabs(Q89) + 2 * r78*fabs(Q78) + 2 * r57*fabs(Q57));

		 dQ_6 = -(c89*r89*Q89*Q89 - c9g*r9g*Q9g*Q9g - cdg*rdg*Qdg*Qdg - cad*rad*Qad*Qad - c8a*r8a*Q8a*Q8a)
			 / (2 * r89*fabs(Q89) + 2 * r9g*fabs(Q9g) + 2 * rdg*fabs(Qdg) + 2 * rad*fabs(Qad) + 2 * r8a*fabs(Q8a));

		 dQ_7 = -(cad*rad *Qad *Qad + cdc*rdc*Qdc*Qdc - cbc*rbc*Qbc*Qbc - cab*rab*Qab*Qab)
			 / (2 * rad*fabs(Qad) + 2 * rdc*fabs(Qdc) + 2 * rbc*fabs(Qbc) + 2 * rab*fabs(Qab));

		 dQ_8 = -(ceb*reb *Qeb *Qeb + cbc*rbc*Qbc*Qbc + ccf*rcf*Qcf*Qcf - cef*ref*Qef*Qef)
			 / (2 * reb*fabs(Qeb) + 2 * rbc*fabs(Qbc) + 2 * rcf*fabs(Qcf) + 2 * ref*fabs(Qef));

		 dQ_9 = -(cdg*rdg *Qdg *Qdg + cgh*rgh*Qgh*Qgh - cfh*rfh*Qfh*Qfh - ccf*rcf*Qcf*Qcf - cdc*rdc*Qdc*Qdc)
			 / (2 * rdg*fabs(Qdg) + 2 * rgh*fabs(Qgh) + 2 * rfh*fabs(Qfh) + 2 * rcf*fabs(Qcf) + 2 * rdc*fabs(Qdc));

		 ds1 = -(-10 + cS0*rS0 *QS0 *QS0 +c01*r01 *Q01 *Q01 + c16*r16*Q16*Q16 + c69*r69*Q69*Q69 + cEA*rEA*QEA*QEA)
			 / (2 * rS0*fabs(QS0) + 2 * r01*fabs(Q01) + 2 * r16*fabs(Q16) + 2 * r69*fabs(Q69) + 2 * rEA*fabs(QEA));

		 ds2 = -(-10 + cS0*rS0 *QS0 *QS0 + c03*r03 *Q03 *Q03 + c3e*r3e*Q3e*Q3e + cef*ref*Qef*Qef + cfh*rfh*Qfh*Qfh + cEB*rEB*QEB*QEB)
			 / (2 * rS0*fabs(QS0) + 2 * r03*fabs(Q03) + 2 * r3e*fabs(Q3e) + 2 * ref*fabs(Qef) + 2 * rfh*fabs(Qfh) + 2 * rEB*fabs(QEB));

		 Q01 = Q01 + dQ_1 + ds1;
		 Q24 = Q24 + dQ_1 - dQ_3;
		 Q57 = Q57 + dQ_3 - dQ_5;
		 Q16 = Q16 + dQ_2 + ds1;
		 Q25 = Q25 - dQ_2 + dQ_3;
		 Q78 = Q78 + dQ_4 - dQ_5;		 
		 Q12 = Q12 + dQ_1 - dQ_2;
		 Q56 = Q56 - dQ_2 + dQ_5;
		 Q89 = Q89 - dQ_5 + dQ_6;
		 Q03 = Q03 - dQ_1 + ds2;
		 Q3e = Q3e - dQ_4 + ds2;;
		 Q69 = Q69 + dQ_5 + ds1;
		 Q34 = Q34 - dQ_1 + dQ_4;
		 Q47 = Q47 - dQ_3 + dQ_4;
		 

		 Q8a = Q8a +dQ_4 - dQ_6;
		 Qab = Qab + dQ_4 - dQ_7;
		 Qad = Qad + dQ_7 - dQ_6;
		 Qeb = Qeb - dQ_4 + dQ_8;
		 Qbc = Qbc - dQ_7 + dQ_8;
		 Qdc = Qdc + dQ_7 - dQ_9;
		 Qdg = Qdg - dQ_6 + dQ_9;
		 Q9g = Q9g - dQ_6;
		 Qgh = Qgh + dQ_9;
		 Qef = Qef - dQ_8 + ds2;
		 Qcf = Qcf + dQ_8 - dQ_9;
		 Qfh = Qfh - dQ_9 + ds2;
		 QS0 = QS0 + ds1 + ds2;
		 QEA = QEA + ds1;
		 QEB = QEB + ds2;


	}
	
	double P01 = c01*r01* Q01*Q01;
	double P24 = c24*r24*Q24*Q24;
	double P57 = c57*r57*Q57*Q57;
	double P16 = c16*r16*Q16*Q16;
	double P25 = c25*r25*Q25*Q25;
	double P78 = c78*r78*Q78*Q78;
	double P12 = c12*r12*Q12*Q12;
	double P56 = c56*r56*Q56*Q56;
	double P89 = c89*r89*Q89*Q89;
	double P03 = c03*r03*Q03*Q03;
	double P3e = c3e*r3e*Q3e*Q3e;
	double P69 = c69*r69*Q69*Q69;
	double P34 = c34*r34*Q34*Q34;
	double P47 = c47*r47*Q47*Q47;


	double P8a = c8a*r8a*Q8a*Q8a;
	double Pab = cab*rab*Qab*Qab;
	double Pad = cad*rad*Qad*Qad;
	double Peb = ceb*reb*Qeb*Qeb;
	double Pbc = cbc*rbc*Qbc*Qbc;
	double Pdc = cdc*rdc*Qdc*Qdc;
	double Pdg = cdg*rdg*Qdg*Qdg;
	double P9g = c9g*r9g*Q9g*Q9g;
	double Pgh = cgh*rgh*Qgh*Qgh;
	double Pef = cef*ref*Qef*Qef;
	double Pcf = ccf*rcf*Qcf*Qcf;
	double Pfh = cfh*rfh*Qfh*Qfh;
	double PS0 = cS0*rS0*QS0*QS0;
	double PEA = cEA*rEA*QEA*QEA;
	double PEB = cEB*rEB*QEB*QEB;


	std::cout << P01 + P12 + P24 - P34 - P03<<endl;
	std::cout << P16 - P56 - P25 - P12 << endl;
	std::cout << P25 + P57 - P47 - P24 << endl;
	std::cout << P34 + P47 + P78 + P8a + Pab - Peb - P3e << endl;
	std::cout << P56 + P69 - P78 - P89 - P57 << endl;
	std::cout << P89 - P9g - Pdg - Pad -P8a<< endl;
	std::cout << Pad + Pdc - Pbc - Pab << endl;
	std::cout << Peb + Pbc + Pcf - Pef << endl;
	std::cout << Pdg + Pgh - Pfh - Pcf - Pdc << endl;
	std::cout << -10 + PS0 + P01 + P16 + P69 + PEA << endl;
	std::cout << -10 + PS0 + P03 + P3e + Pef + Pfh + PEB << endl;

	return 0;
}

