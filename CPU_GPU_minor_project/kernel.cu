#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include<math.h>

double *d_answer;
double *h_answer;
double *hb_answer;

void Allocate_Memory()
{

	hipError_t Error;
	size_t size = 5 * sizeof(double);
	h_answer = (double*)malloc(size);
	hb_answer = (double*)malloc(size);
	Error=hipMalloc((void**)&d_answer, size);


	printf("cuda malloc error message is %s", hipGetErrorString(Error));
}

void Free_Memory()
{
	if (h_answer) free(h_answer);
	if (d_answer) hipFree(d_answer);
}
void Send_To_Device()
{
	size_t size = 5 * sizeof(double);
	hipError_t Error;
	Error = hipMemcpy(d_answer, h_answer, size, hipMemcpyHostToDevice);
	printf("\nCUDA error(memcpy h_answer->d_answer)=%s\n", hipGetErrorString(Error));
}
void Get_From_Memory()
{
	size_t size = 5 * sizeof(double);
	hipError_t Error;	
	Error = hipMemcpy(hb_answer, d_answer, size, hipMemcpyDeviceToHost);
	printf("\nCUDA error(memcpy d_answer->h_answer)=%s\n", hipGetErrorString(Error));
}

__global__ void hardy_cross(double * ��_a)
{
	double Q12 = 10;
	double Q13 = 0;
	double Q23 = 0;
	double Q24 = 10;
	double Q34 = 0;

	double r12 = 5;
	double r13 = 1;
	double r23 = 1;
	double r24 = 1;
	double r34 = 5;
	double dQ_1;
	double dQ_2;

	int c12 = 0;
	int c13 = 0;
	int c23 = 0;
	int c24 = 0;
	int c34 = 0;


	int no_iteraion = 5;

	for (int i = 0; i < no_iteraion; i++)
	{
		if (Q12 != 0)
			c12 = Q12 / fabs(Q12);
		if (Q13 != 0)
			c13 = Q13 / fabs(Q13);
		if (Q23 != 0)
			c23 = Q23 / fabs(Q23);
		if (Q24 != 0)
			c24 = Q24 / fabs(Q24);
		if (Q34 != 0)
			c34 = Q34 / fabs(Q34);



		dQ_1 = -(c12*r12 *Q12 *Q12 + c23*r23*Q23*Q23 - c13*r13*Q13*Q13) / (2 * r12*fabs(Q12) + 2 * r23*fabs(Q23) + 2 * r13*fabs(Q13));

		dQ_2 = -(c24*r24 *Q24 *Q24 - c23*r23*Q23*Q23 - c34*r34*Q34*Q34) / (2 * r24*fabs(Q24) + 2 * r23*fabs(Q23) + 2 * r34*fabs(Q34));

		Q12 = Q12 + dQ_1;
		Q23 = Q23 + dQ_1 - dQ_2;
		Q13 = Q13 - dQ_1;
		Q24 = Q24 + dQ_2;
		Q34 = Q34 - dQ_2;

	}
	
	

	��_a[0] = Q12;
	��_a[1] = Q13;
	��_a[2] = Q23;
	��_a[3] = Q24;
	��_a[4] = Q34;
	


}
void Launch_hardy_cross() 
{
	hardy_cross << <1, 1 >> > (d_answer);
}

