#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"

double *d_answer;
double *h_answer;

void Allocate_Memory()
{
	size_t size = N * sizeof(float);
	hipError_t Error;
	h_a = (float*)malloc(size);
	h_b = (float*)malloc(size);


	Error = hipMalloc((void**)&d_a, size);

	hipMalloc((void**)&d_answer, 5*sizeof(double));

	printf("cuda malloc error message is %s", hipGetErrorString(Error));
}

void Free_Memory()
{
	//if (h_a) free(h_a);
	//if (d_a) hipFree(d_a);
	//if (h_b) free(h_b);
	//if (d_anew) hipFree(d_anew);
}
void Send_To_Device()
{
	//size_t size = N * sizeof(float);
	//hipError_t Error;
	//Error = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	//printf("\nCUDA error(memcpy h_a->d_a)=%s\n", hipGetErrorString(Error));
}
void Get_From_Memory()
{
	size_t size = N * sizeof(float);
	hipError_t Error;
	Error = hipMemcpy(h_b, d_a, size, hipMemcpyDeviceToHost);
	Error = hipMemcpy(h_answer, d_answer, size, hipMemcpyDeviceToHost);


	printf("\nCUDA error(memcpy d_a->h_b)=%s\n", hipGetErrorString(Error));
}

__global__ void hardy_cross()
{
	double Q12 = 10;
	double Q23 = 0;
	double Q13 = 0;
	double Q24 = 10;
	double Q34 = 0;

	double r12 = 1;
	double r13 = 5;
	double r23 = 1;
	double r24 = 5;
	double r34 = 1;
	double dQ_1;
	double dQ_2;

	int no_iteraion = 5;
	for (int i = 0; i < no_iteraion; i++)
	{

		dQ_1 = -(r12 *Q12 *Q12 + r23*Q23*Q23 - r13*Q13*Q13) / (2 * r12*Q12 + 2 * r23*Q23 + 2 * r13*Q13);

		dQ_2 = -(r24 *Q24 *Q24 - r34*Q34*Q34 - r23*Q23*Q23) / (2 * r24*Q24 + 2 * r34*Q34 + 2 * r23*Q23);

		Q12 = Q12 + dQ_1;
		Q23 = Q23 + dQ_1 - dQ_2;
		Q13 = Q13 - dQ_1;
		Q24 = Q24 + dQ_2;
		Q34 = Q34 - dQ_2;

	}


	d_answer[0] = Q12;
	d_answer[1] = Q13;
	d_answer[2] = Q23;
	d_answer[3] = Q24;
	d_answer[4] = Q34;
}
void Launch_hardy_cross() 
{
	hardy_cross();
}

